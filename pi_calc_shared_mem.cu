
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define THD_PER_BLK 256
#define N 9000000
__global__ void pi_calc(double* aread )
{
        int myid = threadIdx.x+blockIdx.x*blockDim.x;
        double x,step;
	if(myid<N){
        	step = 1.0/(double)N;
        	x=(myid)*step;

        	aread[myid]= 4.0/(1.0+x*x);
	}

}

__global__ void sum_reduce(double *arr, double *sum)
{
	int myid=blockIdx.x*blockDim.x+threadIdx.x;
	int range= THD_PER_BLK/2;
	__shared__ double tmp[THD_PER_BLK];
	tmp[threadIdx.x]=0.0;
	if(myid<N)
	{
		tmp[threadIdx.x] = arr[myid];
		__syncthreads();
		while(range>0)
		{
	     		if(threadIdx.x<range){
				tmp[threadIdx.x] += tmp[threadIdx.x+range];
			}
			range=range/2;
			__syncthreads();
		}
		if(threadIdx.x==0)
		{
			sum[blockIdx.x]=tmp[threadIdx.x];
		}
	
	
	}

}

int main()
{

        double  pi;
        double *sum,*aread, *area_small_d ;

        int size=N*sizeof(double);
        double total=0.0;
        double exe_time;
        struct timeval stop_time, start_time;
	int thds_per_block= THD_PER_BLK;
	int num_blocks = (N/thds_per_block)+1;

        double step = 1.0/(double) N;
        //area = 0.0;

	sum= (double *)malloc(N*sizeof(double));

	hipMalloc(&area_small_d, num_blocks*sizeof(double));

        gettimeofday(&start_time, NULL);

        hipMalloc(&aread, size);


        pi_calc<<< num_blocks,thds_per_block >>>(aread);
	hipDeviceSynchronize();

	sum_reduce<<< num_blocks, thds_per_block >>>(aread, area_small_d);

        hipMemcpy(sum, area_small_d, num_blocks*sizeof(double), hipMemcpyDeviceToHost);

        for(int i=0;i<num_blocks;i++){

                total+=sum[i];
        }

        gettimeofday(&stop_time, NULL);
        exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));

        pi = step*total;
        printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);

}

