
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void my_cudasafe( hipError_t error, char* message)
{
        if(error!=hipSuccess)
        {
                fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error));
                exit(-1);
        }
}



__global__ void arrmul(double* md, double* nd, double* pd, double alpha )
{
        int myid = threadIdx.x+blockIdx.x*blockDim.x;

        pd[myid] = md[myid] + alpha*nd[myid]; 
}

int main()
{
	int size = 400 * sizeof(double);
	double a[400], b[400], c[400], alpha;
	double *md, *nd,*pd;
	int i=0;
	
	alpha = 0.001;

	for(i=0; i<400; i++ )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

        //cudaMemcpy(alpha, alpha, 4 , cudaMemcpyHostToDevice);

        my_cudasafe(hipMalloc(&md, size), "Hi");
        my_cudasafe(hipMemcpy(md, a, size, hipMemcpyHostToDevice),"bye");

        my_cudasafe(hipMalloc(&nd, size),"hii");
        my_cudasafe(hipMemcpy(nd, b, size, hipMemcpyHostToDevice),"hiii");

        my_cudasafe(hipMalloc(&pd, size),"Byee");
	
	dim3   DimGrid(1, 1);
        dim3   DimBlock(400, 1);


        arrmul<<< DimGrid,DimBlock >>>(md,nd,pd,alpha);

        my_cudasafe(hipMemcpy(c, pd, size, hipMemcpyDeviceToHost),"heeee");

        for(i=0; i<400; i++ )
        {
                printf("\t%lf",c[i]);
        }

        hipFree(md);
        hipFree(nd);
        hipFree(pd);

}
