
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>



__global__ void pi_calc(double dx, double* aread )
{
        int myid = threadIdx.x+blockIdx.x*blockDim.x;
        double y,x=0.0;
        x = myid*dx;
        y = sqrt(1-x*x);

        aread[myid]= y*dx;

}

int main()
{
        double area[400], pi;
        double* aread;

        int size=400*sizeof(double);
        double dx,total=0.0;
        double exe_time;
        struct timeval stop_time, start_time;

        dx = 1.0/400;
        //area = 0.0;

        gettimeofday(&start_time, NULL);

        hipMalloc(&aread, size);

        dim3   DimGrid(1, 1);
        dim3   DimBlock(400, 1);


        pi_calc<<< DimGrid,DimBlock >>>(dx,aread);

        hipMemcpy(&area, aread, size, hipMemcpyDeviceToHost);

        for(int i=0;i<400;i++){

                total+=area[i];
        }

        gettimeofday(&stop_time, NULL);
        exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));

        pi = 4.0*total;
        printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);

}

