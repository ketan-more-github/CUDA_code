
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void calc_square(int* md, int* pd)
{

        int myid =  threadIdx.x;

        pd[myid] =(md[myid])*(md[myid]);
}


int main()
{
	int size = 400 * sizeof(int);
	int a[400], aa[400],*md,*pd;
	int i=0;
	

	//Initialize the vectors
	for(i=0; i<400; i++ )
	{
		a[i] = i;
	}

	hipMalloc(&md, size);
        hipMemcpy(md, a, size, hipMemcpyHostToDevice);


        hipMalloc(&pd, size);

        dim3   DimGrid(1, 1);
        dim3   DimBlock(400, 1);


        calc_square<<< DimGrid,DimBlock >>>(md,pd);

        hipMemcpy(aa, pd, size, hipMemcpyDeviceToHost);



	//print the output
	for(i=0; i<400; i++ )
	{
		printf("\t%d",aa[i]);
	}	
}




