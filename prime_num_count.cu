
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 10000
#define THD_PER_BLK 1024
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/

__global__ void prime_calc(int* countarr)
{
        int myid = threadIdx.x+blockIdx.x*blockDim.x;
        int flag;
        if(myid<N){
                flag = 1;
                for(int j=2;j<myid;j++)
            	{
                    if((myid%j) == 0)
                    {
                            flag = 0;
                            break;
                    }
            	}
		if(myid>2){
        		countarr[myid]=flag;
		}

        }	

}
__global__ void count_reduce(int *countarr, int *sum)
{
        int myid=blockIdx.x*blockDim.x+threadIdx.x;
        int range= THD_PER_BLK/2;
        __shared__ int tmp[THD_PER_BLK];
        tmp[threadIdx.x]=0;
        if(myid<N)
        {
                tmp[threadIdx.x] =countarr[myid];
                __syncthreads();
                while(range>0)
                {
                        if(threadIdx.x<range){
                                tmp[threadIdx.x] += tmp[threadIdx.x+range];
                        }
                        range=range/2;
                        __syncthreads();
                }
                if(threadIdx.x==0)
                {
                        sum[blockIdx.x]=tmp[threadIdx.x];
                }


        }

}


int main()
{
	int *countarr, *sum, *sum_d;
	double exe_time;
	int thds_per_block= THD_PER_BLK;
        int num_blocks = (N/thds_per_block)+1;
	int size=N*sizeof(int);
	sum= (int *)malloc(N*sizeof(int));
	struct timeval stop_time, start_time;
	
	int count = 1; // 2 is prime. Our loop starts from 3
	
	gettimeofday(&start_time, NULL);
	
	/*
	for(i=3;i<N;i++)
	{
	 	flag = 0;
		for(j=2;j<i;j++)	
	    {
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    }
        if(flag == 0)
        {
        	count++;
        }
	}
	*/
	hipMalloc(&sum_d, num_blocks*sizeof(int));
	hipMalloc(&countarr, size);

	prime_calc<<< num_blocks,thds_per_block >>>(countarr);
	hipDeviceSynchronize();
	
	count_reduce<<< num_blocks, thds_per_block >>>(countarr, sum_d);

	hipMemcpy(sum, sum_d, num_blocks*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i=0;i<num_blocks;i++){

                count+=sum[i];
		//printf("%d ",sum[i]);
        }

	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
	
}
