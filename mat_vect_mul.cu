
#include <hip/hip_runtime.h>
#include"stdio.h"
#include"stdlib.h"


#define N 100

__global__ void arradd(int* a, int* b, int* c)
{
        int myid = blockIdx.x*blockDim.x + threadIdx.x;
	if(myid<N){
		for(int j=0; j<N; j++){

			c[myid] += a[myid*N+j] * b[j];
		}
	}
}



int main(int argc, char **argv)
{
        int *A, *B, *C;
	int i,j;
        int *a,*b, *c;
        int flag = 0;

        B = (int *) malloc(N*sizeof(int));
        A = (int *) malloc(N*N*sizeof(int));
       	C = (int *) malloc(N*sizeof(int));
       	for(i=0;i<N;i++)
        {
            for(j=0;j<N;j++)
            {
                 A[i*N+j] = 1;
            }
                B[i] = 1;
                C[i] = 0;
        }
      

        hipMalloc(&a, N*N*sizeof(int));
        hipMemcpy(a, A, N*N*sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&b, N*sizeof(int));
        hipMemcpy(b, B, N*sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&c, N*sizeof(int));

        dim3   DimGrid(1, 1);
        dim3   DimBlock(100, 1);

        arradd<<< DimGrid,DimBlock >>>(a,b,c);
	
        hipMemcpy(C, c, N*sizeof(int), hipMemcpyDeviceToHost);

	/*
        for(i=0;i<N/size;i++)
        {
            c[i] = 0;
            for(j=0;j<N;j++)
            {
                c[i] += a[i*N+j] * B[j];
            }
        }
	*/

        
       for(i=0;i<N;i++)
       {
          if(C[i] != N)
          {
              flag = 1;
              printf("\n %d", C[i]);
              break;
          }
      }

	if(flag == 0)
        {
              printf("\n PASS!\n");
        }
        else
        {
             printf("\n FAIL!\n");
        }

}

